
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

// mem limited to ~40kB per block -> 256 x len < 40kB, len < 128!
// TODO: make this smarter
#define THREAD_PER_BLOCK 256
#define MAX_PATTERN_LENGTH 16
#define MAX_BLOCK_PER_GRID 65535

int MAX_PATTERN_LENGTH_GPU = MAX_PATTERN_LENGTH;

__device__ int levenshtein(char *s1, char *s2, int len, int *column)
{
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++)
    {
        column[y] = y;
    }
    for (x = 1; x <= len; x++)
    {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++)
        {
            olddiag = column[y];
            column[y] = MIN3(
                column[y] + 1,
                column[y - 1] + 1,
                lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
            lastdiag = olddiag;
        }
    }
    return (column[len]);
}

__global__ void compute_matches_kernel(char *buf, int start, int end, int n_bytes, int length, char *pattern, int approx_factor, int *n_matches)
{
    __shared__ int column[MAX_PATTERN_LENGTH * THREAD_PER_BLOCK];
    int *my_column = &column[threadIdx.x * MAX_PATTERN_LENGTH];
    int j;
    int skip_size = blockDim.x * gridDim.x;
    for (j = start + blockIdx.x * blockDim.x + threadIdx.x; j < end; j += skip_size)
    {
        int distance = 0;
        int size;

        size = length;
        if (n_bytes - j < length)
        {
            size = n_bytes - j;
        }

        distance = levenshtein(pattern, &buf[j], size, my_column);

        if (distance <= approx_factor)
        {
            atomicAdd(n_matches, 1);
        }
    }
}

extern "C" void compute_matches_gpu(char *buf, int start, int end, int n_bytes, char **patterns, int starti, int endi, int approx_factor, int max_pattern_length, int *n_matches)
{
    // shifts the buffer and patterns to the start position
    buf = buf + start;
    end = end - start;
    n_bytes = n_bytes - start;
    start = 0;
    patterns = patterns + starti;
    n_matches = n_matches + starti;
    endi = endi - starti;
    starti = 0;

    int i;
    /* Allocate & transfer */
    char *d_buf;
    int *d_n_matches;
    hipMalloc((void **)&d_buf, sizeof(char) * n_bytes);
    hipMalloc((void **)&d_n_matches, sizeof(int) * endi);
    hipMemcpy(d_buf, buf, sizeof(char) * n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_n_matches, n_matches, sizeof(int) * endi, hipMemcpyHostToDevice);

    /* Traverse the patterns */
    for (i = starti; i < endi; i++)
    {
        int length = strlen(patterns[i]);
        char *pattern = patterns[i];
        char *d_pattern;
        hipMalloc((void **)&d_pattern, sizeof(char) * length);
        hipMemcpy(d_pattern, pattern, sizeof(char) * length, hipMemcpyHostToDevice);

        int block_size = THREAD_PER_BLOCK;
        int num_blocks = (end - start + block_size - 1) / block_size;
        if (num_blocks > MAX_BLOCK_PER_GRID)
        {
            num_blocks = MAX_BLOCK_PER_GRID;
        }
        compute_matches_kernel<<<num_blocks, block_size>>>(d_buf, start, end, n_bytes, length, d_pattern, approx_factor, d_n_matches + i);

        hipFree(d_pattern);
    }

    /* Transfer back */
    hipMemcpy(n_matches, d_n_matches, sizeof(int) * endi, hipMemcpyDeviceToHost);
    /* Free */
    hipFree(d_buf);
    hipFree(d_n_matches);

    hipDeviceSynchronize();
}

__global__ void sum_all_kernel(int *a, int *sum, int n)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? a[i] : 0;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        sum[blockIdx.x] = sdata[0];
    }
}

extern "C" int sum_all(int *a, int n)
{
    int *d_a, *d_sum;
    int sum = 0;

    hipMalloc((void **)&d_a, sizeof(int) * n);
    hipMalloc((void **)&d_sum, sizeof(int) * n);

    hipMemcpy(d_a, a, sizeof(int) * n, hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;

    sum_all_kernel<<<num_blocks, block_size, block_size * sizeof(int)>>>(d_a, d_sum, n);

    while (num_blocks > 1)
    {
        int threads = block_size;
        int blocks = (num_blocks + threads - 1) / threads;
        sum_all_kernel<<<blocks, threads, threads * sizeof(int)>>>(d_sum, d_sum, num_blocks);
        num_blocks = blocks;
    }

    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_sum);

    return sum;
}