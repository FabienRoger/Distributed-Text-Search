
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

// mem limited to ~40kB per block -> 256 x len < 40kB, len < 128!
// TODO: make this smarter
#define MAX_BLOCK_PER_GRID 65535
#define MAX_PATTERN_LENGTH 16

int THREAD_PER_BLOCK;

__device__ int levenshtein(char *s1, char *s2, int len, int *column)
{
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++)
    {
        column[y] = y;
    }
    for (x = 1; x <= len; x++)
    {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++)
        {
            olddiag = column[y];
            column[y] = MIN3(
                column[y] + 1,
                column[y - 1] + 1,
                lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
            lastdiag = olddiag;
        }
    }
    return (column[len]);
}

__global__ void compute_matches_kernel(char *buf, int start, int end, int n_bytes, int length, char *pattern, int approx_factor, int *n_matches, int max_pattern_length)
{
    extern __shared__ int column[];
    int *my_column = &column[threadIdx.x * max_pattern_length];
    int j;
    int skip_size = blockDim.x * gridDim.x;
    for (j = start + blockIdx.x * blockDim.x + threadIdx.x; j < end; j += skip_size)
    {
        int distance = 0;
        int size;

        size = length;
        if (n_bytes - j < length)
        {
            size = n_bytes - j;
        }

        distance = levenshtein(pattern, &buf[j], size, my_column);

        if (distance <= approx_factor)
        {
            atomicAdd(n_matches, 1);
        }
    }
}

extern "C" void compute_matches_gpu(char *buf, int start, int end, int n_bytes, char **patterns, int starti, int endi, int approx_factor, int max_pattern_length, int *n_matches)
{
    // shifts the buffer and patterns to the start position
    buf = buf + start;
    end = end - start;
    n_bytes = n_bytes - start;
    start = 0;
    patterns = patterns + starti;
    n_matches = n_matches + starti;
    endi = endi - starti;
    starti = 0;

    int i;
    /* Allocate & transfer */
    char *d_buf;
    int *d_n_matches;

    hipSetDevice(0);
    hipMalloc((void **)&d_buf, sizeof(char) * n_bytes);
    hipMalloc((void **)&d_n_matches, sizeof(int) * endi);
    hipMemcpy(d_buf, buf, sizeof(char) * n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_n_matches, n_matches, sizeof(int) * endi, hipMemcpyHostToDevice);

    /* Traverse the patterns */
    for (i = starti; i < endi; i++)
    {
        int length = strlen(patterns[i]);
        char *pattern = patterns[i];
        char *d_pattern;
        hipMalloc((void **)&d_pattern, sizeof(char) * length);
        hipMemcpy(d_pattern, pattern, sizeof(char) * length, hipMemcpyHostToDevice);

        int block_size = THREAD_PER_BLOCK;
        int num_blocks = (end - start + block_size - 1) / block_size;
        if (num_blocks > MAX_BLOCK_PER_GRID)
        {
            num_blocks = MAX_BLOCK_PER_GRID;
        }
        compute_matches_kernel<<<num_blocks, block_size, MAX_PATTERN_LENGTH * THREAD_PER_BLOCK * sizeof(int)>>>(d_buf, start, end, n_bytes, length, d_pattern, approx_factor, d_n_matches + i, max_pattern_length);

        hipFree(d_pattern);
    }

    /* Transfer back */
    hipMemcpy(n_matches, d_n_matches, sizeof(int) * endi, hipMemcpyDeviceToHost);
    /* Free */
    hipFree(d_buf);
    hipFree(d_n_matches);

    hipDeviceSynchronize();
}

extern "C" int big_enough_gpu_available(int max_pattern_length)
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (deviceCount < 1)
    {
        return false;
    }
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int max_shared_memory = prop.sharedMemPerBlock;
    int required_shared_memory = max_pattern_length * THREAD_PER_BLOCK * sizeof(int);

    return required_shared_memory < max_shared_memory;
}