
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))
#define MIN2(a, b) (a < b ? a : b)
#define MAX2(a, b) (a > b ? a : b)

// mem limited to ~40kB per block -> 256 x len < 40kB, len < 128!

int THREAD_PER_BLOCK, BLOCK_PER_GRID;
int MAX_BLOCK_PER_GRID, MAX_THREAD_PER_BLOCK, MAX_SHARED_MEMORY_PER_BLOCK; // min of the physical and dictated values
int gpu_initialized = 0;

__device__ int levenshtein(char *s1, char *s2, int len, int *column)
{
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++)
    {
        column[y] = y;
    }
    for (x = 1; x <= len; x++)
    {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++)
        {
            olddiag = column[y];
            column[y] = MIN3(
                column[y] + 1,
                column[y - 1] + 1,
                lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
            lastdiag = olddiag;
        }
    }
    return (column[len]);
}

__global__ void compute_matches_kernel(char *buf, int start, int end, int n_bytes, int length, char *pattern, int approx_factor, int *n_matches, int max_pattern_length)
{
    extern __shared__ int column[];
    int *my_column = &column[threadIdx.x * (max_pattern_length + 1)];
    int j;
    int skip_size = blockDim.x * gridDim.x;
    for (j = start + blockIdx.x * blockDim.x + threadIdx.x; j < end; j += skip_size)
    {
        int distance = 0;
        int size;

        size = length;
        if (n_bytes - j < length)
        {
            size = n_bytes - j;
        }

        distance = levenshtein(pattern, &buf[j], size, my_column);

        if (distance <= approx_factor)
        {
            atomicAdd(n_matches, 1);
        }
    }
}

void initialize_gpu()
{
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    MAX_BLOCK_PER_GRID = MIN2(prop.maxGridSize[0], BLOCK_PER_GRID);
    MAX_THREAD_PER_BLOCK = MIN2(prop.maxThreadsPerBlock, THREAD_PER_BLOCK);
    MAX_SHARED_MEMORY_PER_BLOCK = prop.sharedMemPerBlock;
    gpu_initialized = 1;
}

int required_mem(int length)
{
    return (length + 1) * sizeof(int);
}

extern "C" void compute_matches_gpu(char *buf, int start, int end, int n_bytes, char **patterns, int starti, int endi, int approx_factor, int max_pattern_length, int *n_matches)
{

    if (gpu_initialized == 0)
    {
        initialize_gpu();
    }

    // shifts the buffer and patterns to the start position
    buf = buf + start;
    end = end - start;
    n_bytes = n_bytes - start;
    start = 0;
    patterns = patterns + starti;
    n_matches = n_matches + starti;
    endi = endi - starti;
    starti = 0;

    int i;
    /* Allocate & transfer */
    char *d_buf;
    int *d_n_matches;

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipMallocAsync((void **)&d_buf, sizeof(char) * n_bytes, stream);
    hipMallocAsync((void **)&d_n_matches, sizeof(int) * endi, stream);
    hipMemcpyAsync(d_buf, buf, sizeof(char) * n_bytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_n_matches, n_matches, sizeof(int) * endi, hipMemcpyHostToDevice, stream);

    /* Traverse the patterns */
    for (i = starti; i < endi; i++)
    {
        int length = strlen(patterns[i]);
        char *pattern = patterns[i];
        char *d_pattern;
        hipMallocAsync((void **)&d_pattern, sizeof(char) * length, stream);
        hipMemcpyAsync(d_pattern, pattern, sizeof(char) * length, hipMemcpyHostToDevice, stream);

        int mem_per_thread = required_mem(length);
        int block_size = MIN2(MAX_SHARED_MEMORY_PER_BLOCK / mem_per_thread, MAX_THREAD_PER_BLOCK);
        block_size = MAX2(1, block_size);
        int num_blocks = MIN2((end - start + block_size - 1) / block_size, MAX_BLOCK_PER_GRID);
        num_blocks = MAX2(1, num_blocks);

        compute_matches_kernel<<<num_blocks, block_size, block_size * mem_per_thread, stream>>>(d_buf, start, end, n_bytes, length, d_pattern, approx_factor, d_n_matches + i, length);

        hipFreeAsync(d_pattern, stream);
    }

    /* Transfer back */
    hipMemcpyAsync(n_matches, d_n_matches, sizeof(int) * endi, hipMemcpyDeviceToHost, stream);
    /* Free */
    hipFreeAsync(d_buf, stream);
    hipFreeAsync(d_n_matches, stream);

    // cudaDeviceSynchronize();
}

extern "C" int big_enough_gpu_available(int max_pattern_length)
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (deviceCount < 1)
    {
        return false;
    }
    if (gpu_initialized == 0)
    {
        initialize_gpu();
    }

    int required_shared_memory = required_mem(max_pattern_length);

    return required_shared_memory < MAX_SHARED_MEMORY_PER_BLOCK;
}

extern "C" void sync_gpu()
{
    hipDeviceSynchronize();
}